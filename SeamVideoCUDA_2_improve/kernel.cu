#include "hip/hip_runtime.h"
#include <opencv2\opencv.hpp>
#include <opencv2\highgui\highgui.hpp>
#include <opencv2\core\core.hpp>
#include <opencv2\nonfree\nonfree.hpp>
#include <opencv2\legacy\legacy.hpp>
#include <stdio.h>
#include <ctime>
#include <omp.h>

#include <opencv2\gpu\gpu.hpp>
#include <hip/hip_runtime.h>
#include ""
#include <opencv2/gpu/stream_accessor.hpp>
#include "GetHomography.h"
#include "StitchAndSeam.h"
using namespace std;
using namespace cv;
using namespace cv::gpu;

__global__ void seam_gpu_kernel(const PtrStepSz<uchar3> src1, const PtrStepSz<uchar3> src2, PtrStep<uchar3> dst, const int* seamline)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//int offset = x + y * blockDim.x * gridDim.x;

	if (x < src1.cols && y < src1.rows)
	{
		//uchar3 v = src1(y, x);
		uchar3 v;
		if (x < seamline[y])
			v = src1(y, x);
		else
			v = src2(y, x);
		dst(y, x) = make_uchar3(v.x, v.y, v.z);
	}
}


void seam_gpu_caller(const PtrStepSz<uchar3>& src1, const PtrStepSz<uchar3>& src2, PtrStep<uchar3> dst, const int* seamline, hipStream_t stream)
{
	dim3 block(32, 8);
	dim3 grid((src1.cols + block.x - 1) / block.x, (src1.rows + block.y - 1) / block.y);

	seam_gpu_kernel << <grid, block, 0, stream >> >(src1, src2, dst, seamline);
	if (stream == 0)
		hipDeviceSynchronize();
}


void seam_gpu(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, const vector<int>& gseamLine, Stream& stream = Stream::Null())
{
	dst.create(src1.size(), src1.type());
	hipStream_t s = StreamAccessor::getStream(stream);

	int sz = gseamLine.size();
	int* seam = new int[sz];

//#pragma omp parallel for
	for (int i = 0; i < sz; i++)
		seam[i] = gseamLine[i];
	int* seamline;
	hipMalloc((void**)&seamline, sz*sizeof(int));
	hipMemcpy(seamline, seam, sz*sizeof(int), hipMemcpyHostToDevice);

	seam_gpu_caller(src1, src2, dst, seamline, s);

	hipFree(seamline);
	free(seam);
}